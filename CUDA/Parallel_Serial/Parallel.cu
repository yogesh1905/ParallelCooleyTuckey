
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <cmath>
#include <dirent.h>
#include <cstring>
#include<omp.h>
#include <math.h>
#include <time.h>

using namespace std;

#define N_REPEAT 3

// Complex numbers data type
typedef float2 Cplx;

// Complex numbers operations
static __device__ __host__ inline Cplx CplxAdd(Cplx a, Cplx b) {
  Cplx c; c.x = a.x + b.x; c.y = a.y + b.y; return c;
}

static __device__ __host__ inline Cplx CplxInv(Cplx a) {
  Cplx c; c.x = -a.x; c.y = -a.y; return c;
}

static __device__ __host__ inline Cplx CplxMul(Cplx a, Cplx b) {
  Cplx c; c.x = a.x * b.x - a.y + b.y; c.y = a.x * b.y + a.y * b.x; return c;
}

/**
 * Reorders array by bit-reversing the indexes.
 */
__global__ void bitrev_reorder(Cplx* __restrict__ r, Cplx* __restrict__ d, int s, size_t nthr) {
  int id = blockIdx.x * nthr + threadIdx.x;
  r[__brev(id) >> (32 - s)] = d[id];
}

/**
 * Inner part of FFT loop. Contains the procedure itself.
 */
__device__ void inplace_fft_inner(Cplx* __restrict__ r, int j, int k, int m, int n) {
  if (j + k + m / 2 < n) { 
    Cplx t, u;
    
    t.x = __cosf((2.0 * M_PI * k) / (1.0 * m));
    t.y = -__sinf((2.0 * M_PI * k) / (1.0 * m));
    
    u = r[j + k];
    t = CplxMul(t, r[j + k + m / 2]);

    r[j + k] = CplxAdd(u, t);
    r[j + k + m / 2] = CplxAdd(u, CplxInv(t));
  }
}

/**
 * Middle part of FFT for small scope paralelism.
 */
__global__ void inplace_fft(Cplx* __restrict__ r, int j, int m, int n, size_t nthr) {
  int k = blockIdx.x * nthr + threadIdx.x;
  inplace_fft_inner(r, j, k, m, n);
}

/**
 * Outer part of FFT for large scope paralelism.
 */
__global__ void inplace_fft_outer(Cplx* __restrict__ r, int m, int n, size_t nthr) {
  int j = (blockIdx.x * nthr + threadIdx.x) * m;
  
  for (int k = 0; k < m / 2; k++) {
    inplace_fft_inner(r, j, k, m, n);
  }
}

/**
 * Runs in-place Iterative Fast Fourier Transformation.
 */
void fft(Cplx* __restrict__ d, size_t n, size_t threads, int balance) {
  size_t data_size = n * sizeof(Cplx);
  Cplx *r, *dn;
  
  // Copy data to GPU
  hipMalloc((void**)&r, data_size);
  hipMalloc((void**)&dn, data_size);
  hipMemcpy(dn, d, data_size, hipMemcpyHostToDevice);
  
  // Bit-reversal reordering
  int temp=n;
  int s=0;
  while(temp>0)
  {
    temp/=2;
    s++;
  }
  // int s = log2(n);
  bitrev_reorder<<<ceil(n / threads), threads>>>(r, dn, s, threads);
  
  // Synchronize
  hipDeviceSynchronize();
  
  // Iterative FFT (with loop paralelism balancing)
  for (int i = 1; i <= s; i++) {
    int m = 1 << i;
    if (n/m > balance) {
      inplace_fft_outer<<<ceil((float)n / m / threads), threads>>>(r, m, n, threads);
    } else {
      for (int j = 0; j < n; j += m) {
        float repeats = m / 2;
        inplace_fft<<<ceil(repeats / threads), threads>>>(r, j, m, n, threads);
      }
    }
  }
  
  // Copy data from GPU & free the memory blocks
  Cplx* result;
  result = (Cplx*)malloc(data_size / 2);
  hipMemcpy(result, r, data_size / 2, hipMemcpyDeviceToHost);
  hipFree(r);
  hipFree(dn);
}


int main(int argc, char** argv) {
  srand (time(NULL));
  int n;
  cin>>n;
  int len=pow(2,n);
  vector<Cplx> buffer;
  Cplx temp;
  
  for (int i = 0; i < len; ++i)
    {
      temp.x=i;
      temp.y=i+2;
      buffer.push_back(temp);
    }
  int threads=512;
  int bal=128;
  ofstream myfile;
  // string fname="parallel"+".txt";
  myfile.open("parallel.txt", std::ios_base::app);
  int ind=0;
  for (int z = 1; z <= len; z <<= 1) {
        
      ind+=1;
      // float start = omp_get_wtime();
      // Run FFT algorithm with loaded data
      clock_t t; 
      t = clock(); 
      
      fft(&buffer[0], z, threads, bal);
      
      t = clock() - t; 
      double time_taken = ((double)t)/CLOCKS_PER_SEC;
      myfile <<ind<<" "<<time_taken<<endl;
      cout<<"for n "<<z<<"time " << time_taken <<" seconds"<< endl; 
        
    }
    myfile.close();


  return 0;
}
